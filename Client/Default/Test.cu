#include "hip/hip_runtime.h"
#include"Test.cuh"
#include"hip/hip_runtime.h"
#include<iostream>
#include<hipfft/hipfft.h>
#include"hipblas.h"
#include<stdio.h>
#include<stdlib.h>

CGPUACC::CGPUACC()
{

}

CGPUACC::~CGPUACC()
{

}

__global__ void sum_kernel(int a, int b, int* c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a + b;
}

int CGPUACC::sum_cuda(int a, int b, int* c)
{
	int* f;

	hipMalloc((void**)&f, sizeof(int) * 1);
	hipMemcpy(f, c, sizeof(int) * 1, hipMemcpyHostToDevice);

	sum_kernel <<<1, 1 >>> (a, b, f);

	hipMemcpy(c, f, sizeof(int) * 1, hipMemcpyDeviceToHost);

	hipFree(f);

	return true;
}
